#include "hip/hip_runtime.h"
#include "dist_gpu.cuh"
#include <cmath>
#include <hip/hip_runtime.h>
#include <iostream>
#include <vector>
#include "fmt/core.h"

__global__ void calculate_visibility_kernel(
    const int16_t *height_map,
    unsigned int *visibility_map,
    int width,
    int height,
    int radius,
    int num_angles,
    int y_offset,
    float *ray_directions_x,
    float *ray_directions_y
)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = (blockIdx.y * blockDim.y + threadIdx.y) + y_offset;

    // check bounds
    if (x >= width || y >= height) { return; }

    const int radius_squared = radius * radius;
    const int index = y * width + x;
    unsigned short current_height = height_map[index];

    // Start the count at this cell as 1 (the pixel itself is always visible)
    unsigned int visible_count = 1;

    // Cast rays in different directions
    for (int i = 0; i < num_angles; ++i) {
        float dx = ray_directions_x[i];
        float dy = ray_directions_y[i];

        int max_steps = radius;
        float max_angle_seen = -INFINITY;

        // Step size for more efficient ray traversal
        const float ray_length = sqrtf(dx * dx + dy * dy);
        const float step_x = dx / ray_length;
        const float step_y = dy / ray_length;

        float curr_x_f = x + 0.5f; // Start at center of pixel
        float curr_y_f = y + 0.5f;

        for (int step = 1; step <= max_steps; ++step) {
            // Move along the ray
            curr_x_f += step_x;
            curr_y_f += step_y;

            // Round to nearest pixel
            const int curr_x = __float2int_rn(curr_x_f);
            const int curr_y = __float2int_rn(curr_y_f);

            // Check bounds
            if (curr_x < 0 || curr_x >= width || curr_y < 0 || curr_y >= height) break;

            // Check if we've gone too far (outside the radius)
            int dist_squared = (curr_x - x) * (curr_x - x) + (curr_y - y) * (curr_y - y);
            if (dist_squared > radius_squared) break;

            // Get height at current position
            unsigned short point_height = height_map[curr_y * width + curr_x];

            // Calculate angle to determine visibility
            float distance = sqrtf(static_cast<float>(dist_squared));
            float height_diff = static_cast<float>(point_height) - static_cast<float>(current_height);
            float angle = height_diff / distance;

            // If the angle is greater than the maximum seen so far,
            // the pixel is visible
            if (angle > max_angle_seen) {
                max_angle_seen = angle;
                visible_count++;
            }
        }
    }

    // Store the visibility count
    visibility_map[index] = visible_count;
}

std::vector<unsigned int> calculate_visibility_cuda(
    const std::vector<int16_t> &height_map,
    size_t width,
    size_t height,
    int radius,
    int angle,
    const int start_y, 
    const int end_y, 
    const int my_rank
)
{
    // Calculate the width and height of the visibility map for this process
    const auto my_height = end_y - start_y;
    const auto my_y_offset = start_y;

    // Allocate host result for this process
    std::vector<unsigned int> visibility_map(width * my_height, ~0);

    // Number of discrete angles
    const int num_angles = std::abs(angle);
    // The distance between each angle in radians
    const double angle_step = 2 * M_PI / num_angles;

    // Allocate and initialize host ray directions
    std::vector<float> ray_directions_x(num_angles);
    std::vector<float> ray_directions_y(num_angles);

    // Precalculate the angle of the rays to be cast
    for (int i = 0; i < num_angles; ++i) {
        const double angle_ = i * angle_step;
        ray_directions_x[i] = static_cast<float>(std::cos(angle_) * radius);
        ray_directions_y[i] = static_cast<float>(std::sin(angle_) * radius);
    }

    // device memory
    int16_t *d_height_map = nullptr;
    unsigned int *d_visibility_map = nullptr;
    float *d_ray_directions_x = nullptr;
    float *d_ray_directions_y = nullptr;

    // size calculations
    size_t height_map_size = width * height * sizeof(int16_t);
    size_t visibility_map_size = width * my_height * sizeof(unsigned int);
    size_t ray_directions_size = num_angles * sizeof(float);

    hipMalloc(&d_height_map, height_map_size);
    hipMalloc(&d_visibility_map, visibility_map_size);
    hipMalloc(&d_ray_directions_x, ray_directions_size);
    hipMalloc(&d_ray_directions_y, ray_directions_size);

    // Copy data to device
    hipMemcpy(d_height_map, height_map.data(), height_map_size, hipMemcpyHostToDevice);
    hipMemcpy(d_ray_directions_x, ray_directions_x.data(), ray_directions_size, hipMemcpyHostToDevice);
    hipMemcpy(d_ray_directions_y, ray_directions_y.data(), ray_directions_size, hipMemcpyHostToDevice);

    // Set up grid and block dimensions
    dim3 block_size(16, 16);
    dim3 grid_size((width + block_size.x - 1) / block_size.x, (my_height + block_size.y - 1) / block_size.y);

    // Launch kernel
    std::cout << "Launching CUDA kernel with grid size: " << grid_size.x << "x" << grid_size.y
              << ", block size: " << block_size.x << "x" << block_size.y
              << " from process " << my_rank << std::endl;

    fmt::println("width {} height {} radius {} num_angles {} my_y_offset {}", width, height, radius, num_angles, my_y_offset);
    calculate_visibility_kernel<<<grid_size, block_size>>>(
        d_height_map, d_visibility_map, 
        width, height, radius, num_angles, my_y_offset, 
        d_ray_directions_x, d_ray_directions_y
    );

    // check for errors
    hipError_t error = hipGetLastError();
    if (error != hipSuccess) {
        std::cerr << "CUDA kernel launch failed with error: " << hipGetErrorString(error) << std::endl;
    }


    // wait for kernel to finish
    hipDeviceSynchronize();

    // copy result back to host
    hipMemcpy(visibility_map.data(), d_visibility_map, visibility_map_size, hipMemcpyDeviceToHost);

    // Free device memory
    hipFree(d_height_map);
    hipFree(d_visibility_map);
    hipFree(d_ray_directions_x);
    hipFree(d_ray_directions_y);

    // print a sum of the visibility_map
    unsigned int sum = 0;
    for (unsigned int val : visibility_map) {
        sum += val;
    }
    std::cout << "Sum of visibility map on process " << my_rank << ": " << sum << std::endl;


    std::cout << "CUDA completed on process " << my_rank << std::endl;
    
    return visibility_map;
}
