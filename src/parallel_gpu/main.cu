#include "core.hpp"
#include "parallel_gpu.cuh"
#include <algorithm>
#include <cmath>
#include <cstdlib>
#include <cstring>
#include <hip/hip_runtime.h>
#include <fmt/core.h>
#include <fstream>
#include <iostream>
#include <iterator>
#include <string>
#include <vector>

int main(int argc, char **argv)
{

    std::vector<int16_t> height_map = read_input(argv[1]);
    const size_t width = std::stoul(argv[3]);
    const size_t height = std::stoul(argv[4]);
    const int angle = std::stoi(argv[5]);

    timer time;
    time.reset();

    int radius = 100;
    std::vector<unsigned int> visibility_map = calculate_visibility_cuda(height_map, width, height, radius, angle);

    fmt::println("Elapsed time: {} ms", time.read());

    write_output<unsigned int>(argv[2], visibility_map);

    return 0;
}
